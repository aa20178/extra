#include "hip/hip_runtime.h"
// Includes
#include <iostream>

// includes CUDA
#include <hip/hip_runtime.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>

// Code GPU
__global__ void copymat_device(const float* A, float* B)
{
	// A compléter
}


// Code CPU
void genmat(float *A, int n)
{
	for (int i=0; i<n; i++)
		for (int j=0; j<n; j++)
			A[i*n + j] = rand()/(float) RAND_MAX;
}


float verify(const float *A, const float* B, int n)
{
	float error = 0;
	for (int i=0; i<n; i++)
		for (int j=0; j<n; j++)
			error = std::max(error, abs(A[i*n + j] - B[i*n + j]));

	return error;
}


int main(int argc, char** argv)
{
	printf("Copie de  matrice\n");
	int n = 4096;
	size_t size = n*n*sizeof(float);

	// Matrices CPU
	float *h_A = nullptr, *h_B = nullptr;
	// Matrices GPU
	float *d_A = nullptr, *d_B = nullptr;

	// Allocatation des vecteurs dans la mémoire CPU
	h_A = new float[n];
	h_B = new float[n];

	// Allocation des vecteurs dans la mémoire GPU
	// A compléter

	// Initialisation de la matrice A
	genmat(h_A, n);

	// Copie de la matrice A dans la mémoire GPU
	checkCudaErrors(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

	// Definition de la taille des blocs et de la grille
	dim3 threadsPerBlock(// A compléter);
	dim3 numBlocks(// A compléter);

	copymat_device<<<numBlocks, threadsPerBlock>>>(d_A, d_B);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

	// Copie du résultat
	checkCudaErrors(hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost));

	// Verification
	printf("Erreur max: %e\n", verify(h_A, h_B, n));

	// Timing
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	const int nb = 10;
	checkCudaErrors(hipEventRecord(start, 0));
	for (int i = 0; i < nb; i++)
		copymat_device<<<numBlocks, threadsPerBlock>>>(d_A, d_B);
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	float t_ms;
	checkCudaErrors(hipEventElapsedTime(&t_ms, start, stop));
	t_ms /= nb;

	printf("Temps d'exécution du Kernel : %e (ms)\n", t_ms);
	printf("Bande passante GPU: %e GO/s\n", // A compléter);

	// Deallocation de la memoire GPU
	// A compléter

	// Deallocation de la memoire CPU
	delete [] h_a;
	delete [] h_b;
}
